#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "displacement_op_cuda.h"
#include "stdio.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ double norm(double3 v) {
  return sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
}

__device__ int3 get_box_coordinates(double3 pos, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords;
  box_coords.x = (floor(pos.x) - grid_dimensions[0]) / box_length;
  box_coords.y = (floor(pos.y) - grid_dimensions[1]) / box_length;
  box_coords.z = (floor(pos.z) - grid_dimensions[2]) / box_length;
  return box_coords;
}

__device__ uint32_t get_box_id_2(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0]*num_boxes_axis[1] + bc.y * num_boxes_axis[0] + bc.x;
}

__device__ uint32_t get_box_id(double3 pos, uint32_t* num_boxes_axis, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length);
  return get_box_id_2(box_coords, num_boxes_axis);
}

__device__ void compute_force(double* positions, double* diameters, uint32_t idx, uint32_t nidx, double3* result) {
  double r1 = 0.5 * diameters[idx];
  double r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a desired density.
  double additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  double comp1 = positions[3*idx + 0] - positions[3*nidx + 0];
  double comp2 = positions[3*idx + 1] - positions[3*nidx + 1];
  double comp3 = positions[3*idx + 2] - positions[3*nidx + 2];
  double center_distance = sqrt(comp1 * comp1 + comp2 * comp2 + comp3 * comp3);

  // the overlap distance (how much one penetrates in the other)
  double delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    result->x += 42.0;
    result->y += 42.0;
    result->z += 42.0;
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  double r = (r1 * r2) / (r1 + r2);
  double gamma = 1; // attraction coeff
  double k = 2;     // repulsion coeff
  double f = k * delta - gamma * sqrt(r * delta);

  double module = f / center_distance;
  result->x += module * comp1;
  result->y += module * comp2;
  result->z += module * comp3;
}

__device__ void default_force(double* positions,
                   double* diameters,
                   uint32_t idx, uint32_t start, uint16_t length,
                   uint32_t* successors,
                   double3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    // implement logic for within radius here
    if (nidx != idx) {
      compute_force(positions, diameters, idx, nidx, result);
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__global__ void collide(
       double* positions,
       double* diameters,
       double* tractor_force,
       double* adherence,
       double* mass,
       double* timestep,
       double* max_displacement,
       uint32_t* N,
       uint32_t* starts,
       uint16_t* lengths,
       uint32_t* successors,
       uint32_t* box_length,
       uint32_t* num_boxes_axis,
       int32_t* grid_dimensions,
       double* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < N[0]) {
		result[3*tidx + 0] += timestep[0] * tractor_force[3*tidx + 0];
    result[3*tidx + 1] += timestep[0] * tractor_force[3*tidx + 1];
    result[3*tidx + 2] += timestep[0] * tractor_force[3*tidx + 2];
    
    double3 pos;
    pos.x = positions[3*tidx + 0];
    pos.y = positions[3*tidx + 1];
    pos.z = positions[3*tidx + 2];

    double3 collision_force = make_double3(0, 0, 0);

    // Moore neighborhood
    int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length[0]);
    for (int z = -1; z <= 1; z++) {
      for (int y = -1; y <= 1; y++) {
        for (int x = -1; x <= 1; x++) {
          uint32_t bidx = get_box_id_2(box_coords + make_int3(x, y, z), num_boxes_axis);
          if (lengths[bidx] != 0) {
            default_force(positions, diameters, tidx, starts[bidx], lengths[bidx], successors, &collision_force);
          }
        }
      }
    }

    // Mass needs to non-zero!
    double mh = timestep[0] / mass[tidx];

    if (norm(collision_force) > adherence[tidx]) {
      result[3*tidx + 0] += collision_force.x * mh;
      result[3*tidx + 1] += collision_force.y * mh;
      result[3*tidx + 2] += collision_force.z * mh;

      if (norm(collision_force) * mh > max_displacement[0]) {
        result[3*tidx + 0] = max_displacement[0];
        result[3*tidx + 1] = max_displacement[0];
        result[3*tidx + 2] = max_displacement[0];
      }
    }
  }
}

void displacement_op_cuda(double* positions, double* diameters, double* tractor_force, double* adherence, double* mass, double* timestep, double* max_displacement, uint32_t* N, uint32_t* starts, uint16_t* lengths, uint32_t* successors, uint32_t* box_length, uint32_t* num_boxes_axis, int32_t* grid_dimensions, double* cell_movements) {
	double* d_positions = NULL;
    double* d_diameters = NULL;
    double* d_mass = NULL;
    double* d_timestep = NULL;
    double* d_max_displacement = NULL;
    uint32_t* d_N = NULL;
    double* d_cell_movements = NULL;
    double* d_tractor_force = NULL;
    double* d_adherence = NULL;
    uint32_t* d_starts = NULL;
    uint16_t* d_lengths = NULL;
    uint32_t* d_successors = NULL;
    uint32_t* d_box_length = NULL;
    uint32_t* d_num_boxes_axis = NULL;
    int32_t* d_grid_dimensions = NULL;

    uint32_t num_boxes = num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

    hipMalloc(&d_positions, 3 * N[0] * sizeof(double));
    hipMalloc(&d_diameters, N[0] * sizeof(double));
    hipMalloc(&d_tractor_force, 3 * N[0] * sizeof(double));
    hipMalloc(&d_adherence, N[0] * sizeof(double));
    hipMalloc(&d_mass, N[0] * sizeof(double));
    hipMalloc(&d_timestep, sizeof(double));
    hipMalloc(&d_max_displacement, sizeof(double));
    hipMalloc(&d_N, sizeof(uint32_t));
    hipMalloc(&d_starts, num_boxes * sizeof(uint32_t));
    hipMalloc(&d_lengths, num_boxes * sizeof(uint16_t));
    hipMalloc(&d_successors, N[0] * sizeof(uint32_t));
    hipMalloc(&d_box_length, sizeof(uint32_t));
    hipMalloc(&d_num_boxes_axis, 3 * sizeof(uint32_t));
    hipMalloc(&d_grid_dimensions, 3 * sizeof(int32_t));
    hipMalloc(&d_cell_movements, 3 * N[0] * sizeof(double));

    hipMemcpy(d_positions, 		positions, 3 * N[0] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_diameters, 		diameters, N[0] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_tractor_force, 	tractor_force, 3 * N[0] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_adherence, 		adherence, N[0] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, 				mass, N[0] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_timestep, 			timestep, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_max_displacement, 	max_displacement, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_N, 				N, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_starts, 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice);
    gpuErrchk(hipMemcpy(d_lengths, 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_successors, 		successors, N[0] * sizeof(uint32_t), hipMemcpyHostToDevice));
    hipMemcpy(d_box_length, 		box_length, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_num_boxes_axis, 	num_boxes_axis, 3 * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_dimensions, 	grid_dimensions, 3 * sizeof(uint32_t), hipMemcpyHostToDevice);

    int blockSize;
    int minGridSize;
    int gridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0, N[0]);
    gridSize = (N[0] + blockSize - 1) / blockSize;

    printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
    collide<<<gridSize, blockSize>>>(d_positions, d_diameters, d_tractor_force, d_adherence, d_mass, d_timestep, d_max_displacement, d_N, d_starts, d_lengths, d_successors, d_box_length, d_num_boxes_axis, d_grid_dimensions, d_cell_movements);

    hipDeviceSynchronize();
    hipMemcpy(cell_movements, d_cell_movements, 3 * N[0] * sizeof(double), hipMemcpyDeviceToHost);
}
